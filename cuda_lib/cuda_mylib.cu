#include "hip/hip_runtime.h"
#include "cuda_mylib.h"

#include <stdio.h>

void SayHi() {
  printf("Hello Cuda!\n");
}

// CUDA device code
__global__ void
vectorAdd(const float *A, const float *B, float *C, int size) {
    int i = 1;//blockDim.x * blockIdx.x + threadIdx.x;

    //if (i <= size) {
        C[i] = 0;//size;//A[i] + B[i];
    //}
}

void CheckCUDAError(const hipError_t& err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA failed: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// C++ host code
void Add(const float *A, const float *B, float *C, int size) {
  if (size <= 0) {
    return;
  }
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);
  CheckCUDAError(err);

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);
  CheckCUDAError(err);

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);
  CheckCUDAError(err);
  
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  CheckCUDAError(err);
  err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  CheckCUDAError(err);
  err = hipMemcpy(d_C, B, size, hipMemcpyHostToDevice);
  CheckCUDAError(err);
  
  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 64;
  int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", 
         blocksPerGrid, threadsPerBlock);
         
  // CUDA code called here
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
  err = hipGetLastError();
  CheckCUDAError(err);
    
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  CheckCUDAError(err);
    
  CheckCUDAError(hipFree(d_A));
  CheckCUDAError(hipFree(d_B));
  CheckCUDAError(hipFree(d_C));
}