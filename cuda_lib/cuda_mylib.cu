#include "hip/hip_runtime.h"
#include "cuda_mylib.h"

#include <stdio.h>

void SayHi() {
  printf("Hello Cuda!\n");
}

// CUDA device code
__global__ void 
vectorAdd(float* A, float* B, float* C, int size) {
    int i = blockIdx.x * blockDim.x  + threadIdx.x; 
    if (i < size) {
        C[i] = A[i] + B[i];
    }
}

void CheckCUDAError(const hipError_t& err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA failed: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// C++ host code
void Add(const float *A, const float *B, float *C, int size) {
  if (size <= 0) {
    return;
  }
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size*sizeof(float));
  CheckCUDAError(err);

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size*sizeof(float));
  CheckCUDAError(err);

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size*sizeof(float));
  CheckCUDAError(err);
  
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
  CheckCUDAError(err);
  err = hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);
  CheckCUDAError(err);
  err = hipMemcpy(d_C, B, size * sizeof(float), hipMemcpyHostToDevice);
  CheckCUDAError(err);
  
  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 64;
  int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", 
         blocksPerGrid, threadsPerBlock);
         
  // CUDA code called here
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
  err = hipGetLastError();
  CheckCUDAError(err);
    
  hipDeviceSynchronize();

  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);
  CheckCUDAError(err);
    
  CheckCUDAError(hipFree(d_A));
  CheckCUDAError(hipFree(d_B));
  CheckCUDAError(hipFree(d_C));
}